#include "hip/hip_runtime.h"
#include "VtClothSolverGPU.cuh"
#include "Common.cuh"
#include "Common.hpp"
#include "Timer.hpp"

// ����ʹ�ã�������꣬�м�*****ע��*****


using namespace std;

namespace Velvet
{
	// constant ָ�������ڴ棬����GPU��������ķ��ʷ�ʽ�������ṩ����Ķ�ȡ����
	// ���ǣ���Щ����������CUDA�����ִ�й����У����ǲ����޸ĵģ������Ա���ȡ
	__device__ __constant__ VtSimParams d_params;
	VtSimParams h_params;

	__device__ inline void AtomicAdd(glm::vec3* address, int index, glm::vec3 val, int reorder)
	{
		// reorder��֤��ԭ�Ӳ�������ȷ��
		/*
		���Խ�����֤��ʹ����reorder֮��r1��r2��r3��ֵΨһ�����������������㣩
		*/
		int r1 = reorder % 3;
		int r2 = (reorder + 1) % 3;
		int r3 = (reorder + 2) % 3;
		// ʵ��������ֵ�ķ���������ۼӲ���
		atomicAdd(&(address[index].x) + r1, val[r1]);
		atomicAdd(&(address[index].x) + r2, val[r2]);
		atomicAdd(&(address[index].x) + r3, val[r3]);
	}

	// ����ģ�����ʱ��
	void SetSimulationParams(VtSimParams* hostParams)
	{
		ScopedTimerGPU timer("Solver_SetParams");
		// checkCudaErrors���CUDA�����ķ���ֵ�������κδ���
		// hipMemcpyToSymbolAsync ���ڽ����ݴ������ڴ��첽���Ƶ��豸�ĳ����ڴ�
		/*
		�������ڴ��е�����ͨ���첽�������Ƶ��豸�ĳ����ڴ�"d_params"�У�
		�����豸�ϵĺ�������ʹ�ó����ڴ��е����ݽ��м��㣬������Ҫ�������ڴ��ȡ���ݣ��Ӷ��������
		*/
		checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_params), hostParams, sizeof(VtSimParams)));
		h_params = *hostParams;
	}

	// ��ʼ��λ��
	__global__ void InitializePositions_Kernel(glm::vec3* positions, const int start, const int count, const glm::mat4 modelMatrix)
	{
		GET_CUDA_ID(id, count);
		// ��λ��ת��������������
		positions[start + id] = modelMatrix * glm::vec4(positions[start+id], 1);
	}

	void InitializePositions(glm::vec3* positions, const int start, const int count, const glm::mat4 modelMatrix)
	{
		ScopedTimerGPU timer("Solver_Initialize");
		CUDA_CALL(InitializePositions_Kernel, count)(positions, start, count, modelMatrix);
	}

	// Ԥ��λ��  ���������position�ǲ��ܸ��ĵģ�����ΪCONST
	__global__ void PredictPositions_Kernel(
		glm::vec3* predicted,
		glm::vec3* velocities,
		CONST(glm::vec3*) positions,
		const float deltaTime)
	{
		/*
		�ж��ٸ����ӣ��ʹ������ٸ��̣߳�
		*/
		GET_CUDA_ID(id, d_params.numParticles);

		//glm::vec3 gravity = glm::vec3(0, -10, 0);
		// �����ٶ�
		velocities[id] += d_params.gravity * deltaTime;
		// ����λ��
		predicted[id] = positions[id] + velocities[id] * deltaTime;
	}

	void PredictPositions(
		glm::vec3* predicted, 
		glm::vec3* velocities,
		CONST(glm::vec3*) positions,
		const float deltaTime)
	{
		ScopedTimerGPU timer("Solver_Predict");
		CUDA_CALL(PredictPositions_Kernel, h_params.numParticles)(predicted, velocities, positions, deltaTime);
	}

	// ��������Լ�� ( deltas��ʵ����delta_p )
	__global__ void SolveStretch_Kernel(
		glm::vec3* predicted,
		glm::vec3* deltas,
		int* deltaCounts,
		CONST(int*) stretchIndices,
		CONST(float*) stretchLengths,
		CONST(float*) invMasses,
		const uint numConstraints)
	{
		// �ж���Լ�����ʹ��������߳�
		GET_CUDA_ID(id, numConstraints);

		int idx1 = stretchIndices[2 * id];
		int idx2 = stretchIndices[2 * id + 1];
		// Ԥ�ھ���
		float expectedDistance = stretchLengths[id];

		glm::vec3 diff = predicted[idx1] - predicted[idx2];
		float distance = glm::length(diff);
		// ���������ĵ���
		float w1 = invMasses[idx1];
		float w2 = invMasses[idx2];
		// ������벻������������ && �������������������Ϊ0
		// �൱�����ĳ���Ǿ�ֹ�㣬���������������ر��Ϳ�����
		if (distance != expectedDistance && w1 + w2 > 0)
		{
			// ���漸�ж�ӦPBD���ĵĹ�ʽ(9)(10)
			glm::vec3 gradient = diff / (distance + EPSILON);
			// compliance is zero, therefore XPBD=PBD
			float denom = w1 + w2;
			float lambda = (distance - expectedDistance) / denom;
			glm::vec3 common = lambda * gradient;
			glm::vec3 correction1 = -w1 * common;
			glm::vec3 correction2 = w2 * common;


			int reorder = idx1 + idx2;
			// ���ǽ�correction1�ĸ����������ӵ�delats��
			AtomicAdd(deltas, idx1, correction1, reorder);
			AtomicAdd(deltas, idx2, correction2, reorder);
			/*
			�����д������ڼ���Լ��Ӧ�õĴ�����ÿ�μ���Լ��ʱdeltaCounts�����ж�Ӧ����ļ�����������1
			�ü������������ں�����Լ����������㷨����Ϊ  ����ݼ�������ֵ�����ɳ����ӻ��������
			*/
			atomicAdd(&deltaCounts[idx1], 1);
			atomicAdd(&deltaCounts[idx2], 1);
			//printf("correction[%d] = (%.2f,%.2f,%.2f)\n", idx1, correction1.x, correction1.y, correction1.z);
			//printf("correction[%d] = (%.2f,%.2f,%.2f)\n", idx2, correction2.x, correction2.y, correction2.z);
		}
	}

	void SolveStretch(
		glm::vec3* predicted,
		glm::vec3* deltas,
		int* deltaCounts,
		CONST(int*) stretchIndices, 
		CONST(float*) stretchLengths,
		CONST(float*) invMasses,
		const uint numConstraints)
	{
		ScopedTimerGPU timer("Solver_SolveStretch");
		CUDA_CALL(SolveStretch_Kernel, numConstraints)(predicted, deltas, deltaCounts, stretchIndices, stretchLengths, invMasses, numConstraints);
	}

	// ��������Լ��
	__global__ void SolveBending_Kernel(
		glm::vec3* predicted,
		glm::vec3* deltas,
		int* deltaCounts,
		CONST(uint*) bendingIndices,
		CONST(float*) bendingAngles,
		CONST(float*) invMass,
		const uint numConstraints,
		const float deltaTime)
	{
		// ����Լ����������ID
		GET_CUDA_ID(id, numConstraints);
		uint idx1 = bendingIndices[id * 4];
		uint idx2 = bendingIndices[id * 4+1];
		uint idx3 = bendingIndices[id * 4+2];
		uint idx4 = bendingIndices[id * 4+3];
		// ���������Ƕ�
		float expectedAngle = bendingAngles[id];
		// ��������
		float w1 = invMass[idx1];
		float w2 = invMass[idx2];
		float w3 = invMass[idx3];
		float w4 = invMass[idx4];

		glm::vec3 p1 = predicted[idx1];
		glm::vec3 p2 = predicted[idx2] - p1;
		glm::vec3 p3 = predicted[idx3] - p1;
		glm::vec3 p4 = predicted[idx4] - p1;
		glm::vec3 n1 = glm::normalize(glm::cross(p2, p3));
		glm::vec3 n2 = glm::normalize(glm::cross(p2, p4));

		float d = clamp(glm::dot(n1, n2), 0.0f, 1.0f);
		float angle = acos(d);
		// cross product for two equal vector produces NAN
		if (angle < EPSILON || isnan(d)) return;

		glm::vec3 q3 = (glm::cross(p2, n2) + glm::cross(n1, p2) * d) / (glm::length(glm::cross(p2, p3)) + EPSILON);
		glm::vec3 q4 = (glm::cross(p2, n1) + glm::cross(n2, p2) * d) / (glm::length(glm::cross(p2, p4)) + EPSILON);
		glm::vec3 q2 = -(glm::cross(p3, n2) + glm::cross(n1, p3) * d) / (glm::length(glm::cross(p2, p3)) + EPSILON)
			- (glm::cross(p4, n1) + glm::cross(n2, p4) * d) / (glm::length(glm::cross(p2, p4)) + EPSILON);
		glm::vec3 q1 = -q2 - q3 - q4;

		float xpbd_bend = d_params.bendCompliance / deltaTime / deltaTime;
		// ��ӦCSDN bending ��https://blog.csdn.net/weixin_43940314/article/details/129830991��
		float denom = xpbd_bend + (w1 * glm::dot(q1, q1) + w2 * glm::dot(q2, q2) + w3 * glm::dot(q3, q3) + w4 * glm::dot(q4, q4));
		if (denom < EPSILON) return; // ?
		float lambda = sqrt(1.0f - d * d) * (angle - expectedAngle) / denom;

		int reorder = idx1 + idx2 + idx3 + idx4;
		AtomicAdd(deltas, idx1, w1 * lambda * q1, reorder);
		AtomicAdd(deltas, idx2, w2 * lambda * q2, reorder);
		AtomicAdd(deltas, idx3, w3 * lambda * q3, reorder);
		AtomicAdd(deltas, idx4, w4 * lambda * q4, reorder);
		
		atomicAdd(&deltaCounts[idx1], 1);
		atomicAdd(&deltaCounts[idx2], 1);
		atomicAdd(&deltaCounts[idx3], 1);
		atomicAdd(&deltaCounts[idx4], 1);
	}

	void SolveBending(
		glm::vec3* predicted,
		glm::vec3* deltas,
		int* deltaCounts,
		CONST(uint*) bendingIndices,
		CONST(float*) bendingAngles,
		CONST(float*) invMass,
		const uint numConstraints,
		const float deltaTime)
	{
		ScopedTimerGPU timer("Solver_SolveBending");
		CUDA_CALL(SolveBending_Kernel, numConstraints)(predicted, deltas, deltaCounts, bendingIndices, bendingAngles, invMass, numConstraints, deltaTime);
	}

	// ������Լ��
	/*
	����Լ��������ģ��������ӡ���������������Ч��������ʵ�ָ�������µĸ��Ź�ϵ
	����̶������һ���֡�ά������֮������ӡ�����������α��
	*/
	__global__ void SolveAttachment_Kernel(
		glm::vec3* predicted,
		glm::vec3* deltas,
		int* deltaCounts,
		CONST(float*) invMass,
		CONST(int*) attachParticleIDs,
		CONST(int*) attachSlotIDs,
		CONST(glm::vec3*) attachSlotPositions,
		CONST(float*) attachDistances,
		const int numConstraints)
	{
		GET_CUDA_ID(id, numConstraints);

		uint pid = attachParticleIDs[id];
		// ��ȡ���Ų�λλ��
		glm::vec3 slotPos = attachSlotPositions[attachSlotIDs[id]];
		// d_params.longRangeStretchiness����������� ��������Ŀ������������
		float targetDist = attachDistances[id] * d_params.longRangeStretchiness;
		if (invMass[pid] == 0 && targetDist > 0) return;

		glm::vec3 pred = predicted[pid];
		glm::vec3 diff = pred - slotPos;
		float dist = glm::length(diff);
		// ��Ҫ����
		if (dist > targetDist)
		{
			//float coefficient = max(targetDist, dist - 0.1*d_params.particleDiameter);// 0.05 * targetDist + 0.95 * dist;
			glm::vec3 correction = -diff + diff / dist * targetDist;
			AtomicAdd(deltas, pid, correction, id);
			atomicAdd(&deltaCounts[pid], 1);
		}
	}

	void SolveAttachment(
		glm::vec3* predicted,
		glm::vec3* deltas,
		int* deltaCounts,
		CONST(float*) invMass,
		CONST(int*) attachParticleIDs,
		CONST(int*) attachSlotIDs,
		CONST(glm::vec3*) attachSlotPositions,
		CONST(float*) attachDistances,
		const int numConstraints)
	{
		ScopedTimerGPU timer("Solver_SolveAttach");
		CUDA_CALL(SolveAttachment_Kernel, numConstraints)(predicted, deltas, deltaCounts, 
			invMass, attachParticleIDs, attachSlotIDs, attachSlotPositions, attachDistances, numConstraints);
	}

	// Ӧ�������� deltas
	__global__ void ApplyDeltas_Kernel(glm::vec3* predicted, glm::vec3* deltas, int* deltaCounts)
	{
		GET_CUDA_ID(id, d_params.numParticles);

		float count = (float)deltaCounts[id];
		// Լ����������0
		if (count > 0)
		{
			// Ӧ�øı���
			predicted[id] += deltas[id] / count * d_params.relaxationFactor;
			// �ı�����Ϊ0
			deltas[id] = glm::vec3(0);
			// �������
			deltaCounts[id] = 0;
		}
	}

	void ApplyDeltas(glm::vec3* predicted, glm::vec3* deltas, int* deltaCounts)
	{
		ScopedTimerGPU timer("Solver_ApplyDeltas");
		CUDA_CALL(ApplyDeltas_Kernel, h_params.numParticles)(predicted, deltas, deltaCounts);
	}

	// ����Ħ��������������
	__device__ glm::vec3 ComputeFriction(glm::vec3 correction, glm::vec3 relVel,glm::vec3 friction1)
	{
		glm::vec3 friction = glm::vec3(0);
		// ��������������ģ
		float correctionLength = glm::length(correction);
		if (d_params.friction > 0 && correctionLength > 0)
		{
			// ��������������λ����
			glm::vec3 norm = correction / correctionLength;

			glm::vec3 tanVel = relVel - norm * glm::dot(relVel, norm);
			float tanLength = glm::length(tanVel);
			float maxTanLength = correctionLength * d_params.friction;

			friction = -tanVel * min(maxTanLength / tanLength, 1.0f)* friction1;
		}
		return friction;
	}

	__global__ void CollideSDF_Kernel(
		glm::vec3* predicted,			// Ԥ��λ��
		CONST(SDFCollider*) colliders,	// ��ײ��
		CONST(glm::vec3*) positions,	// λ��
		const uint numColliders,		// ��ײ������
		const float deltaTime)			// ʱ����
	{
		// �������������������߳�
		GET_CUDA_ID(id, d_params.numParticles);
		// ��ǰλ�ã����������ɣ�
		auto pos = positions[id];
		// Ԥ��λ��
		auto pred = predicted[id];
		for (int i = 0; i < numColliders; i++)
		{
			auto collider = colliders[i];
			// ������������
			glm::vec3 correction = collider.ComputeSDF(pred, d_params.collisionMargin);
			// Ӧ����������
			pred += correction;

			if (glm::dot(correction, correction) > 0)
			{
				//����Ħ����
				glm::vec3 friction1 = glm::vec3(5.0);
				// ��������ٶ�  Ԥ��λ�� - ��ǰλ�� - ��ײ֮���ٶ� * ʱ����
				glm::vec3 relVel = pred - pos - collider.VelocityAt(pred) * deltaTime;
				auto friction = ComputeFriction(correction, relVel,friction1);
				pred += friction;
			}
		}
		predicted[id] = pred;
	}

	void CollideSDF(
		glm::vec3* predicted,
		CONST(SDFCollider*) colliders,
		CONST(glm::vec3*) positions,
		const uint numColliders,
		const float deltaTime)
	{
		ScopedTimerGPU timer("Solver_CollideSDFs");
		if (numColliders == 0) return;
		
		CUDA_CALL(CollideSDF_Kernel, h_params.numParticles)(predicted, colliders, positions, numColliders, deltaTime);
	}
	
	// ��ײ����������
	__global__ void CollideParticles_Kernel(
		glm::vec3* deltas,
		int* deltaCounts,
		CONST(glm::vec3*) predicted,
		CONST(float*) invMasses,
		CONST(uint*) neighbors,
		CONST(glm::vec3*) positions)
	{
		// �����������������߳�
		GET_CUDA_ID(id, d_params.numParticles);

		glm::vec3 positionDelta = glm::vec3(0);
		int deltaCount = 0;
		glm::vec3 pred_i = predicted[id];
		glm::vec3 vel_i = (pred_i - positions[id]);
		float w_i = invMasses[id];

		for (int neighbor = id; neighbor < d_params.numParticles * d_params.maxNumNeighbors; neighbor += d_params.numParticles)
		{
			uint j = neighbors[neighbor];
			if (j > d_params.numParticles) break;

			float w_j = invMasses[j];
			float denom = w_i + w_j;
			if (denom <= 0) continue;

			glm::vec3 pred_j = predicted[j];
			glm::vec3 diff = pred_i - pred_j;
			float distance = glm::length(diff);
			if (distance >= d_params.particleDiameter) continue;

			glm::vec3 gradient = diff / (distance + EPSILON);
			float lambda = (distance - d_params.particleDiameter) / denom;
			glm::vec3 common = lambda * gradient;

			deltaCount++;
			positionDelta -= w_i * common;

			glm::vec3 relativeVelocity = vel_i - (pred_j - positions[j]);
			//
			glm::vec3 friction1 = glm::vec3(5.0);
			glm::vec3 friction = ComputeFriction(common, relativeVelocity, friction1);
			positionDelta += w_i * friction;
		}

		deltas[id] = positionDelta;
		deltaCounts[id] = deltaCount;
	}

	void CollideParticles(
		glm::vec3* deltas,
		int* deltaCounts,
		glm::vec3* predicted,
		CONST(float*) invMasses,
		CONST(uint*) neighbors,
		CONST(glm::vec3*) positions)
	{
		ScopedTimerGPU timer("Solver_CollideParticles");
		// �ȼ�����ײ
		CUDA_CALL(CollideParticles_Kernel, h_params.numParticles)(deltas, deltaCounts, predicted, invMasses, neighbors, positions);
		// Ӧ������
		CUDA_CALL(ApplyDeltas_Kernel, h_params.numParticles)(predicted, deltas, deltaCounts);
	}

	__global__ void Finalize_Kernel(
		glm::vec3* velocities,
		glm::vec3* positions,
		CONST(glm::vec3*) predicted,
		const float deltaTime)
	{
		// ���������������߳�ID
		GET_CUDA_ID(id, d_params.numParticles);
		// ��λ��
		glm::vec3 new_pos = predicted[id];
		// ��λ�ò����ٶ�
		glm::vec3 raw_vel = (new_pos - positions[id]) / deltaTime;
		// �ٶȵ�ģ
		float raw_vel_len = glm::length(raw_vel);
		// �����������ٶȣ���ʹ������ٶȼ���
		if (raw_vel_len > d_params.maxSpeed)
		{
			raw_vel = raw_vel / raw_vel_len * d_params.maxSpeed;
			new_pos = positions[id] + raw_vel * deltaTime;
			//printf("Limit vel[%.3f>%.3f] for id[%d]. Pred[%.3f,%.3f,%.3f], Pos[%.3f,%.3f,%.3f]\n", raw_vel_len, d_params.maxSpeed, id);
			//printf("new_pos %f %f %f\n", new_pos.x, new_pos.y, new_pos.z);
		}
		velocities[id] = raw_vel * (1 - d_params.damping * deltaTime);
		positions[id] = new_pos;
		//printf("new_pos %f %f %f\n", new_pos.x, new_pos.y, new_pos.z);
	}

	// ģ������첽  ִ�иú���(�൱�ڰ����е������������꣬����Ӧ�õ�λ����)
	void Finalize(
		glm::vec3* velocities, 
		glm::vec3* positions,
		CONST(glm::vec3*) predicted,
		const float deltaTime)
	{
		ScopedTimerGPU timer("Solver_Finalize");
		CUDA_CALL(Finalize_Kernel, h_params.numParticles)(velocities, positions, predicted, deltaTime);
	}

	// ���������η���
	__global__ void ComputeTriangleNormals(
		glm::vec3* normals,
		CONST(glm::vec3*) positions,
		CONST(uint*) indices,
		uint numTriangles)
	{
		// �������������������߳���
		GET_CUDA_ID(id, numTriangles);
		uint idx1 = indices[id * 3];
		uint idx2 = indices[id * 3+1];
		uint idx3 = indices[id * 3+2];

		auto p1 = positions[idx1];
		auto p2 = positions[idx2];
		auto p3 = positions[idx3];

		auto normal = glm::cross(p2 - p1, p3 - p1);
		//if (isnan(normal.x) || isnan(normal.y) || isnan(normal.z)) normal = glm::vec3(0, 1, 0);

		int reorder = idx1 + idx2 + idx3;
		AtomicAdd(normals, idx1, normal, reorder);
		AtomicAdd(normals, idx2, normal, reorder);
		AtomicAdd(normals, idx3, normal, reorder);
	}

	// ���㶥�㷨�ߣ���ʵ���ǹ�һ����
	__global__ void ComputeVertexNormals(glm::vec3* normals)
	{
		// �����������������߳���
		GET_CUDA_ID(id, d_params.numParticles);

		auto normal = glm::normalize(normals[id]);
		//normals[id] = glm::vec3(0,1,0);
		normals[id] = normal;
	}

	// ���㷨��
	void ComputeNormal(
		glm::vec3* normals,
		CONST(glm::vec3*) positions, 
		CONST(uint*) indices, 
		const uint numTriangles)
	{
		ScopedTimerGPU timer("Solver_UpdateNormals");
		if (h_params.numParticles)
		{
			// ��ͬ�������ڴ�
			hipMemsetAsync(normals, 0, h_params.numParticles * sizeof(glm::vec3));
			// ��һ�������������η���
			CUDA_CALL(ComputeTriangleNormals, numTriangles)(normals, positions, indices, numTriangles);
			// �ڶ�������һ������
			CUDA_CALL(ComputeVertexNormals, h_params.numParticles)(normals);
		}
	}

}